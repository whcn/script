#include "hip/hip_runtime.h"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/core/utility.hpp"
#include "opencv2/highgui/highgui_c.h"

#include <hip/hip_runtime.h>
/* #include <thrust/count.h> */ 
/* #include <thrust/device_vector.h> */ 

#include <iostream>
#include <tuple> 

using namespace std;
using namespace cv;

#define WIDTH 3

struct pixel {
	/* pixel(){} */
	/* pixel(int red, int green, int blue): r(red), g(green), b(blue){} */
	int r;
	int g;
	int b;
};


__global__ void infaint( int *r, int *g, int *b, int rows, int cols, int *ret ) {

	int offsetX = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetY = blockDim.y * blockIdx.y + threadIdx.y;
	int tid = offsetY * cols + offsetX;

	int tmp[10][10];
	for (int i = 0; i < 10; ++i) {
		for (int j = 0; j < 10; ++j) {
			tmp[i][j] = -2;
		} 
		 
	} 
	int counts[WIDTH * 3 + 1][WIDTH * 3 + 1] = {0};


	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {

			r[i * cols + j] = counts[i][j]+1;
		} 
		 
	} 

}

int main(int argc, char** argv )
{

	int rows = 10;
	int cols = 10;

	int *h_r, *h_g, *h_b;
    int *dev_r, *dev_g, *dev_b;

	int *h_ret;
	int *dev_ret;

    // allocate the memory on the CPU
	h_r = (int*)malloc(rows * cols * sizeof(int));
	h_g = (int*)malloc(rows * cols * sizeof(int));
	h_b = (int*)malloc(rows * cols * sizeof(int));
	h_ret = (int*)malloc(rows * cols * sizeof(int));

	// split Mat into rgb array
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			h_r[i * cols + j] = 1;
			h_g[i * cols + j] = 1;
			h_b[i * cols + j] = 1;

			h_ret[i * cols + j] = -1;
			/* cout << h_r[i * cols + j] << ' ' << h_g[i * cols + j] << ' ' << h_b[i * cols + j] << endl; */
		}  
	} 

	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			cout << h_r[i * cols + j] << ' ';
		}
		cout << endl;
	}

    // allocate the memory on the GPU
	hipError_t t;
    t = hipMalloc( (void**)&dev_r, rows * cols * sizeof(int) );
    t = hipMalloc( (void**)&dev_g, rows * cols * sizeof(int) );
    t = hipMalloc( (void**)&dev_b, rows * cols * sizeof(int) );

    t = hipMalloc( (void**)&dev_ret, rows * cols * sizeof(int) );

    // copy the host memory to device memory
    t = hipMemcpy( dev_r, h_r, rows * cols * sizeof(int), hipMemcpyHostToDevice );
    t = hipMemcpy( dev_g, h_g, rows * cols * sizeof(int), hipMemcpyHostToDevice );
    t = hipMemcpy( dev_b, h_b, rows * cols * sizeof(int), hipMemcpyHostToDevice );

t = hipMemcpy( dev_ret, h_ret, rows * cols * sizeof(int), hipMemcpyHostToDevice );

	dim3 threadsPerBlock(5, 5);
	dim3 blocksPerGrid(2, 2);


    infaint<<<blocksPerGrid, threadsPerBlock>>>( dev_r, dev_g, dev_b, rows, cols, dev_ret );

    // copy the array 'c' back from the GPU to the CPU
    t = hipMemcpy( h_r, dev_r, rows * cols * sizeof(int), hipMemcpyDeviceToHost );
    t = hipMemcpy( h_g, dev_g, rows * cols * sizeof(int), hipMemcpyDeviceToHost );
    t = hipMemcpy( h_b, dev_b, rows * cols * sizeof(int), hipMemcpyDeviceToHost );

t = hipMemcpy( h_ret, dev_ret, rows * cols * sizeof(int), hipMemcpyDeviceToHost );

	if (t != hipSuccess) { 
		cout << "Failed"  << endl;
	} 

	// display image
	cout << endl;
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			cout << h_r[i * cols + j] << ' ';
		}
		cout << endl;
	}

    // free the memory allocated on the CPU
    free( h_r );
    free( h_g );
    free( h_b );

    // free the memory allocated on the GPU
    hipFree( dev_r );
    hipFree( dev_g );
    hipFree( dev_b );

    return 0;
}
