#include "hip/hip_runtime.h"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/core/utility.hpp"
#include "opencv2/highgui/highgui_c.h"

#include <hip/hip_runtime.h>
/* #include <thrust/count.h> */ 
/* #include <thrust/device_vector.h> */ 

#include <iostream>
#include <tuple> 

using namespace std;
using namespace cv;

#define WIDTH 2

struct pixel {
	/* pixel(){} */
	/* pixel(int red, int green, int blue): r(red), g(green), b(blue){} */
	int r;
	int g;
	int b;
};


__global__ void infaint( int *r, int *g, int *b, int rows, int cols, int *ret ) {

	int offsetX = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetY = blockDim.y * blockIdx.y + threadIdx.y;
	int tid = offsetY * cols + offsetX;


	if ((offsetX >= WIDTH) && (offsetX < (cols - WIDTH)) && (offsetY >= WIDTH) && (offsetY < (rows - WIDTH))) { 
		if ((r[tid] == 255) && (g[tid] == 255) && (b[tid] == 255)) { 

/* ret[tid] = offsetX; */

			pixel pixels[2 * WIDTH + 1][2 * WIDTH + 1];

			for (int is = offsetY - WIDTH, i = offsetY - WIDTH; i <= offsetY + WIDTH; ++i) {
				for (int js = offsetX - WIDTH, j = offsetX - WIDTH; j <= offsetX + WIDTH; ++j) {
					int sid = i * cols + j;
					pixel tmp;
					tmp.r = r[sid];
					tmp.g = g[sid];
					tmp.b = b[sid];
					pixels[i - is][j - js] = tmp;
				} 
			} 


			int counts[2 * WIDTH + 1][2 * WIDTH + 1] = {0};

			for (int i = 0; i < 2 * WIDTH + 1; ++i) {
				for (int j = 0; j < 2 * WIDTH + 1; ++j) {
					for (int p = 0; p < 2 * WIDTH + 1; ++p) {
						for (int q = 0; q < 2 * WIDTH + 1; ++q) {
							if ((pixels[i][j].r == pixels[p][q].r) && (pixels[i][j].g == pixels[p][q].g) && (pixels[i][j].b == pixels[p][q].b)) { 
								counts[i][j] += 1;
							} 
						}
					}  
				} 
			} 
	ret[tid] = pixels[0][0].r;



/* 			int ix = WIDTH, iy = WIDTH; */
/* 			int ic = -1; */
/* 			for (int i = 0; i < 2 * WIDTH + 1; ++i) { */
/* 				for (int j = 0; j < 2 * WIDTH + 1; ++j) { */
/* 					if (ic < counts[i][j]) { */ 
/* 						ic = counts[i][j]; */
/* 						ix = i; */
/* 						iy = j; */
/* 					} */ 
/* 				} */
/* 			} */
/* 			r[tid] = pixels[ix][iy].r; */
/* 			g[tid] = pixels[ix][iy].g; */
/* 			b[tid] = pixels[ix][iy].b; */
		} 
	} 
}

int main(int argc, char** argv )
{
	Mat img = imread("./data/extract.jpg", CV_LOAD_IMAGE_UNCHANGED);

	if (img.empty()) { 
		cout << "Load Image Fail!" << endl;
		return -1;
	} 

	/* hipDeviceProp_t devProp; */
	/* hipGetDeviceProperties(&devProp, 0); */
	/* cout << devProp.maxThreadsPerBlock  << endl; */
	/* for (int i = 0; i < 3; ++i) { */
	/* 		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]); */
	/* 		printf("Maximum dimension %d of gird:  %d\n", i, devProp.maxGridSize[i]); */
	/* } */
			

	int rows = img.rows;
	int cols = img.cols;
	int channels = img.channels();

	int *h_r, *h_g, *h_b;
    int *dev_r, *dev_g, *dev_b;

	int *h_ret;
	int *dev_ret;

    // allocate the memory on the CPU
	h_r = (int*)malloc(rows * cols * sizeof(int));
	h_g = (int*)malloc(rows * cols * sizeof(int));
	h_b = (int*)malloc(rows * cols * sizeof(int));
	h_ret = (int*)malloc(rows * cols * sizeof(int));

	// split Mat into rgb array
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			h_r[i * cols + j] = (int)img.at<Vec3b>(i, j)[0];
			h_g[i * cols + j] = (int)img.at<Vec3b>(i, j)[1];
			h_b[i * cols + j] = (int)img.at<Vec3b>(i, j)[2];

			h_ret[i * cols + j] = -1;
			/* cout << h_r[i * cols + j] << ' ' << h_g[i * cols + j] << ' ' << h_b[i * cols + j] << endl; */
		}  
	} 

    // allocate the memory on the GPU
	hipError_t t;
    t = hipMalloc( (void**)&dev_r, rows * cols * sizeof(int) );
    t = hipMalloc( (void**)&dev_g, rows * cols * sizeof(int) );
    t = hipMalloc( (void**)&dev_b, rows * cols * sizeof(int) );

t = hipMalloc( (void**)&dev_ret, rows * cols * sizeof(int) );

    // copy the host memory to device memory
    t = hipMemcpy( dev_r, h_r, rows * cols * sizeof(int), hipMemcpyHostToDevice );
    t = hipMemcpy( dev_g, h_g, rows * cols * sizeof(int), hipMemcpyHostToDevice );
    t = hipMemcpy( dev_b, h_b, rows * cols * sizeof(int), hipMemcpyHostToDevice );

t = hipMemcpy( dev_ret, h_ret, rows * cols * sizeof(int), hipMemcpyHostToDevice );

	dim3 threadsPerBlock(32, 32);
	dim3 blocksPerGrid(15, 15);

	/* cout << h_r[0] << ' ' << h_g[0] << ' ' << h_b[0] << endl; */

    infaint<<<blocksPerGrid, threadsPerBlock>>>( dev_r, dev_g, dev_b, rows, cols, dev_ret );

    // copy the array 'c' back from the GPU to the CPU
    t = hipMemcpy( h_r, dev_r, rows * cols * sizeof(int), hipMemcpyDeviceToHost );
    t = hipMemcpy( h_g, dev_g, rows * cols * sizeof(int), hipMemcpyDeviceToHost );
    t = hipMemcpy( h_b, dev_b, rows * cols * sizeof(int), hipMemcpyDeviceToHost );

t = hipMemcpy( h_ret, dev_ret, rows * cols * sizeof(int), hipMemcpyDeviceToHost );

	if (t != hipSuccess) { 
		cout << "Failed"  << endl;
	} 

	// display image
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			/* img.at<Vec3b>(i, j)[0] = h_r[i * cols + j]; */
			/* img.at<Vec3b>(i, j)[1] = h_g[i * cols + j]; */
			/* img.at<Vec3b>(i, j)[2] = h_b[i * cols + j]; */
			img.at<Vec3b>(i, j)[0] = h_r[i * cols + j];
			img.at<Vec3b>(i, j)[1] = h_g[i * cols + j];
			img.at<Vec3b>(i, j)[2] = h_b[i * cols + j];

		}  
	} 

	for (int i = 0; i < rows * cols; ++i) {
	/* for (int i = 0; i < 10; ++i) { */
		cout << h_ret[i] << ' ';
	} 
	cout << endl;

	namedWindow("image", CV_WINDOW_AUTOSIZE);
	imshow("image", img);
	waitKey(0);
	destroyWindow("image");

    // free the memory allocated on the CPU
    free( h_r );
    free( h_g );
    free( h_b );

    // free the memory allocated on the GPU
    hipFree( dev_r );
    hipFree( dev_g );
    hipFree( dev_b );

    return 0;
}
